#include <iostream>
#include <vector>
#include <cstdlib>
#include <hip/hip_runtime.h>

typedef long long int64;

const int64 mod = 12289;
const int64 root = 11;
const int N = 1024;

// Helper functions for modular arithmetic
__device__ int64 modmul(int64 a, int64 b) {
    return (a * b) % mod;
}

__device__ int64 modadd(int64 a, int64 b) {
    return (a + b) % mod;
}

__device__ int64 modsub(int64 a, int64 b) {
    return (a - b + mod) % mod;
}

__host__ __device__ int64 modpow(int64 base, int64 exp, int64 m) {
    int64 res = 1;
    base %= m;
    while (exp > 0) {
        if (exp & 1) res = res * base % m;
        base = base * base % m;
        exp >>= 1;
    }
    return res;
}

// Kernel for a single stage of NTT
__global__ void ntt_stage_kernel(int64* a, int n, int len, int64 root_power, bool invert) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int group_size = len / 2;
    
    // Calculate which group and position within the group this thread is processing
    int group = idx / group_size;
    int pos = idx % group_size;
    
    // Calculate actual index to process
    int i = group * len + pos;
    
    // Ensure we're within bounds
    if (i + group_size < n) {
        // Calculate twiddle factor
        int64 wlen = root_power;
        if (invert) wlen = modpow(wlen, mod - 2, mod);
        
        int64 w = modpow(wlen, pos, mod);
        
        // Perform butterfly operation
        int64 u = a[i];
        int64 v = modmul(a[i + group_size], w);
        a[i] = modadd(u, v);
        a[i + group_size] = modsub(u, v);
    }
}

// Kernel for pointwise multiplication
__global__ void pointwise_mult_kernel(int64* A, int64* B, int64* C, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        C[i] = modmul(A[i], B[i]);
    }
}

// Kernel for applying inverse NTT scaling factor
__global__ void scale_kernel(int64* a, int n, int64 n_inv) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        a[i] = modmul(a[i], n_inv);
    }
}

// Function to perform complete NTT
void perform_ntt(int64* d_data, int n, bool invert, hipStream_t stream = 0) {
    int threads = 256;
    
    // For each stage of the NTT
    for (int len = 2; len <= n; len <<= 1) {
        int group_size = len / 2;
        int num_groups = n / len;
        int total_threads = num_groups * group_size;
        int blocks = (total_threads + threads - 1) / threads;
        
        // Calculate root power for this stage
        int64 root_power = modpow(root, (mod - 1) / len, mod);
        
        // Launch kernel for this stage
        ntt_stage_kernel<<<blocks, threads, 0, stream>>>(d_data, n, len, root_power, invert);
    }
    
    // Apply scaling factor for inverse NTT
    if (invert) {
        int blocks = (n + threads - 1) / threads;
        int64 n_inv = modpow(n, mod - 2, mod);
        scale_kernel<<<blocks, threads, 0, stream>>>(d_data, n, n_inv);
    }
}

// Function for single GPU implementation
void run_single_gpu(int64* h_A, int64* h_B, int n) {
    hipSetDevice(0);
    
    int64 *d_A, *d_B, *d_C;
    hipStream_t stream;
    hipStreamCreate(&stream);
    
    // Allocate device memory
    hipMalloc(&d_A, n * sizeof(int64));
    hipMalloc(&d_B, n * sizeof(int64));
    hipMalloc(&d_C, n * sizeof(int64));
    
    // Copy input data to device
    hipMemcpyAsync(d_A, h_A, n * sizeof(int64), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_B, h_B, n * sizeof(int64), hipMemcpyHostToDevice, stream);
    
    // Create events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Start timing
    hipEventRecord(start, stream);
    
    // Forward NTT
    perform_ntt(d_A, n, false, stream);
    perform_ntt(d_B, n, false, stream);
    
    // Pointwise multiplication
    int threads = 256;
    int blocks = (n + threads - 1) / threads;
    pointwise_mult_kernel<<<blocks, threads, 0, stream>>>(d_A, d_B, d_C, n);
    
    // Inverse NTT
    perform_ntt(d_C, n, true, stream);
    
    // Stop timing
    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    
    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    // Copy results back to host
    int64* result = new int64[n];
    hipMemcpyAsync(result, d_C, n * sizeof(int64), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
    
    // Print results
    std::cout << "[SINGLE GPU] Result (first 10 coeffs): ";
    for (int i = 0; i < 10; ++i)
        std::cout << result[i] << " ";
    std::cout << "...\n";
    
    std::cout << "[SINGLE GPU] Total GPU time: " << milliseconds << " ms\n";
    
    // Clean up
    delete[] result;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipStreamDestroy(stream);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

// Function for dual GPU implementation
void run_dual_gpu(int64* h_A, int64* h_B, int n) {
    int device_count;
    hipGetDeviceCount(&device_count);
    if (device_count < 2) {
        std::cerr << "Dual GPU implementation requires at least 2 GPUs. Falling back to single GPU." << std::endl;
        run_single_gpu(h_A, h_B, n);
        return;
    }
    
    int half_N = n / 2;
    int64 *d_A0, *d_B0, *d_C0;
    int64 *d_A1, *d_B1, *d_C1;
    
    // Create streams for asynchronous operations
    hipStream_t stream0, stream1;
    
    // Create events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // GPU 0 setup
    hipSetDevice(0);
    hipStreamCreate(&stream0);
    hipMalloc(&d_A0, half_N * sizeof(int64));
    hipMalloc(&d_B0, half_N * sizeof(int64));
    hipMalloc(&d_C0, half_N * sizeof(int64));
    hipMemcpyAsync(d_A0, h_A, half_N * sizeof(int64), hipMemcpyHostToDevice, stream0);
    hipMemcpyAsync(d_B0, h_B, half_N * sizeof(int64), hipMemcpyHostToDevice, stream0);
    
    // GPU 1 setup
    hipSetDevice(1);
    hipStreamCreate(&stream1);
    hipMalloc(&d_A1, half_N * sizeof(int64));
    hipMalloc(&d_B1, half_N * sizeof(int64));
    hipMalloc(&d_C1, half_N * sizeof(int64));
    hipMemcpyAsync(d_A1, h_A + half_N, half_N * sizeof(int64), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(d_B1, h_B + half_N, half_N * sizeof(int64), hipMemcpyHostToDevice, stream1);
    
    // Calculate kernel launch parameters
    int threads = 256;
    int blocks = (half_N + threads - 1) / threads;
    
    // Start timing
    hipSetDevice(0);
    hipEventRecord(start);
    
    // GPU 0: Forward NTT for both arrays
    hipSetDevice(0);
    perform_ntt(d_A0, half_N, false, stream0);
    perform_ntt(d_B0, half_N, false, stream0);
    
    // GPU 1: Forward NTT for both arrays
    hipSetDevice(1);
    perform_ntt(d_A1, half_N, false, stream1);
    perform_ntt(d_B1, half_N, false, stream1);
    
    // GPU 0: Pointwise multiplication
    hipSetDevice(0);
    pointwise_mult_kernel<<<blocks, threads, 0, stream0>>>(d_A0, d_B0, d_C0, half_N);
    
    // GPU 1: Pointwise multiplication
    hipSetDevice(1);
    pointwise_mult_kernel<<<blocks, threads, 0, stream1>>>(d_A1, d_B1, d_C1, half_N);
    
    // GPU 0: Inverse NTT
    hipSetDevice(0);
    perform_ntt(d_C0, half_N, true, stream0);
    
    // GPU 1: Inverse NTT
    hipSetDevice(1);
    perform_ntt(d_C1, half_N, true, stream1);
    
    // Stop timing
    hipSetDevice(0);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    // Copy results back to host
    int64* result = new int64[n];
    
    hipSetDevice(0);
    hipMemcpyAsync(result, d_C0, half_N * sizeof(int64), hipMemcpyDeviceToHost, stream0);
    hipStreamSynchronize(stream0);
    
    hipSetDevice(1);
    hipMemcpyAsync(result + half_N, d_C1, half_N * sizeof(int64), hipMemcpyDeviceToHost, stream1);
    hipStreamSynchronize(stream1);
    
    // Print results
    std::cout << "[DUAL GPU] Result (first 10 coeffs): ";
    for (int i = 0; i < 10; ++i)
        std::cout << result[i] << " ";
    std::cout << "...\n";
    
    std::cout << "[DUAL GPU] Total GPU time: " << milliseconds << " ms\n";
    
    // Clean up
    delete[] result;
    
    hipSetDevice(0);
    hipFree(d_A0);
    hipFree(d_B0);
    hipFree(d_C0);
    hipStreamDestroy(stream0);
    
    hipSetDevice(1);
    hipFree(d_A1);
    hipFree(d_B1);
    hipFree(d_C1);
    hipStreamDestroy(stream1);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main(int argc, char** argv) {
    // Use first argument to determine mode (1 for single GPU, 2 for dual GPU, or both if not specified)
    int mode = 0;
    if (argc > 1) {
        mode = std::atoi(argv[1]);
        if (mode != 1 && mode != 2) {
            std::cerr << "Invalid mode. Use 1 for single GPU or 2 for dual GPU." << std::endl;
            return 1;
        }
    }
    
    // Initialize input data
    int64* h_A = new int64[N];
    int64* h_B = new int64[N];
    
    srand(42);
    for (int i = 0; i < N; ++i) {
        h_A[i] = rand() % mod;
        h_B[i] = rand() % mod;
    }
    
    // Check available GPUs
    int device_count;
    hipGetDeviceCount(&device_count);
    std::cout << "Available GPUs: " << device_count << std::endl;
    
    if (device_count == 0) {
        std::cerr << "No CUDA-capable GPUs found!" << std::endl;
        return 1;
    }
    
    // Run the appropriate benchmark(s)
    if (mode == 0 || mode == 1) {
        std::cout << "\n=== Running Single GPU Benchmark ===\n";
        run_single_gpu(h_A, h_B, N);
    }
    
    if (mode == 0 || mode == 2) {
        if (device_count >= 2) {
            std::cout << "\n=== Running Dual GPU Benchmark ===\n";
            run_dual_gpu(h_A, h_B, N);
        } else {
            std::cout << "\nSkipping Dual GPU benchmark (requires 2 GPUs)\n";
        }
    }
    
    // Clean up host memory
    delete[] h_A;
    delete[] h_B;
    
    return 0;
}
